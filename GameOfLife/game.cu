#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE 25





__host__ __device__ int countAlive (int i, int j, int N, int* in);
__host__ __device__ void check (int index, int *count, int * in);

  /* The Kernel procedure. Assumes that the grid and blocks are squared, so we have a square matrix*/
__global__ void game( int N, int* in, int* out ) {
  int i = blockIdx.y*blockDim.y + threadIdx.y;
  int j = blockIdx.x*blockDim.x + threadIdx.x;
  int neighbours=countAlive(i,j,N,in);

  if ((neighbours==3) || ((neighbours==2) && (in[i*N + j]==1))){
    out [i*N + j]=1;
  }
  else {
    out [i*N + j]=0;
  }
  __syncthreads();
}


__global__ void game2( int N, int* in, int* out, int thperblok ) {
  int i = blockIdx.y*blockDim.y + threadIdx.y;
  int j = blockIdx.x*blockDim.x + threadIdx.x;
  int k=0;
  for (k=0; k<thperblok;k++){
    int neighbours=countAlive(i,j* thperblok + k,N,in);

    if ((neighbours==3) || ((neighbours==2) && (in[i*N + j*thperblok + k ]==1))){
      out [i*N + j*thperblok + k]=1;
    }
    else {
      out [i*N + j*thperblok + k]=0;
    }
  }

  __syncthreads();
}


__global__ void game5(int N, int* in, int* out, int thperblok){

   __shared__ extern int tab[] ;

  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int x =threadIdx.x;
  int y=threadIdx.y;
  int sizeline = (blockDim.x)*thperblok+ 2;
  int k=0;

  for (k=0;k<thperblok;k++){
    if ((x==0) && (y==0)){ //bottom left
      if (k==0){
      tab[k] = in[((i-1+N)%N)*N+((j*thperblok +k-1+N)%N)]; //bottom left
      tab[(y+1)*sizeline+ x*thperblok + k]=in[i*N+((j*thperblok + k-1+N)%N)]; //left
    }
      tab[(y+1)*sizeline + (x+1)*thperblok -(thperblok-1) +k]=in[i*N + j*thperblok + k]; //self
      tab[y*sizeline + (x+1)*thperblok-(thperblok-1)+k]=in[(((i-1+N)%N)*N) + j*thperblok + k]; // down
    }
    else if ((x==0)&&(y!=0)&&(y!=blockDim.y-1)){ //all left
      if (k==0){
      tab[(y+1)*sizeline+ x*thperblok + k]=in[i*N+((j*thperblok + k-1+N)%N)]; //left
    }
      tab[(y+1)*sizeline + (x+1)*thperblok -(thperblok-1) +k]=in[i*N + j*thperblok + k]; //self
    }
    else if ((x==blockDim.x-1)&&(y==0)){ //bottom right
      if (k==(thperblok-1)){
      tab[(blockDim.x+1)*thperblok-(thperblok-1)]=in[((i-1+N)%N)*N+((j*thperblok + k+1)%N)];//bottom right
      tab[(y+1)*sizeline+ (x+2)*thperblok -(thperblok-1)]=in[(i*N) + ((j*thperblok + k+1)%N)];//right
    }
      tab[(y+1)*sizeline + (x+1)*thperblok -(thperblok-1) +k]=in[i*N + j*thperblok + k]; //self
      tab[y*sizeline + (x+1)*thperblok-(thperblok-1)+k]=in[(((i-1+N)%N)*N) + j*thperblok + k]; // down
    }
    else if((x==blockDim.x-1)&&(y!=0)&&(y!=blockDim.y-1)){// all right
      if (k==(thperblok-1)){
      tab[(y+1)*sizeline+ (x+2)*thperblok -(thperblok-1)]=in[(i*N) + ((j*thperblok + k+1)%N)];//right
    }
      tab[(y+1)*sizeline + (x+1)*thperblok -(thperblok-1)+k]=in[i*N + j*thperblok + k]; //self
    }
    else if ((x!=0)&&(x!=blockDim.x-1)&&(y==0)){ // all bottom
      tab[y*sizeline + (x+1)*thperblok-(thperblok-1)+k]=in[(((i-1+N)%N)*N) + j*thperblok + k]; // down 
      tab[(y+1)*sizeline + (x+1)*thperblok -(thperblok-1) +k]=in[i*N + j*thperblok + k]; //self
    }
    else if ((x==blockDim.x-1)&&(y==blockDim.y-1)){//right up
      if (k==(thperblok-1)){
      tab[sizeline*(blockDim.y+2) -1 ]=in[(((i+1)%N)*N) + ((j*thperblok + k+1)%N)];//right up
      tab[(y+1)*sizeline+ (x+2)*thperblok -(thperblok-1)]=in[(i*N) + ((j*thperblok + k+1)%N)];//right
    }
      tab[(y+1)*sizeline + (x+1)*thperblok -(thperblok-1) +k]=in[i*N + j*thperblok + k]; //self
      tab[(y+2)*sizeline + (x+1)*thperblok -(thperblok-1) + k]=in[(((i+1)%N)*N) + j*thperblok + k]; //up
    }
    else if ((x==0)&&(y==blockDim.y-1)){//left up
      if (k==0){
      tab[(y+2)*sizeline + k]=in[(((i+1)%N)*N) + ((j*thperblok + k-1+N)%N)];//up left
      tab[(y+1)*sizeline+ x*thperblok + k]=in[i*N+((j*thperblok + k-1+N)%N)]; //left
    }
      tab[(y+1)*sizeline + (x+1)*thperblok -(thperblok-1) +k]=in[i*N + j*thperblok + k]; //self
      tab[(y+2)*sizeline + (x+1)*thperblok -(thperblok-1) + k]=in[(((i+1)%N)*N) + j*thperblok + k]; //up
    }
    else if ((x!=0)&&(x!=blockDim.x)&&(y==blockDim.y-1)){//all up
      tab[(y+2)*sizeline + (x+1)*thperblok -(thperblok-1) + k]=in[(((i+1)%N)*N) + j*thperblok + k]; //up
      tab[(y+1)*sizeline + (x+1)*thperblok -(thperblok-1) +k]=in[i*N + j*thperblok + k]; //self
    }
    else {
        tab[(y+1)*sizeline + (x+1)*thperblok -(thperblok-1) +k]=in[i*N + j*thperblok + k]; //all other threads that are not on the side of the block
    }
}
  __syncthreads();

  for (k=0;k<thperblok;k++){
      int neighbours = countAlive(y+1,(x+1)*thperblok -(thperblok-1) + k,sizeline, tab);
      if ((neighbours==3) || ((neighbours==2) && (tab[(y+1)*sizeline + (x+1)*thperblok -(thperblok-1) +k]==1))){
        out [i*N + j*thperblok + k]=1;
      }
      else {
        out [i*N + j*thperblok + k]=0;
      }
  }

}


__host__ __device__ int countAlive (int i, int j, int N, int* array){
  int count=0;



  check ( (((i+1)%N)*N) + j, &count, array); //up cell
  check ( (((i-1+N)%N)*N) + j, &count, array); //down cell
  check ( (i*N) + ((j+1)%N), &count, array); //right cell
  check ( (i*N) + ((j-1+N)%N), &count, array); //left cell
  check ( (((i+1)%N)*N) + ((j+1)%N), &count, array); //right up cell
  check ( (((i+1)%N)*N) + ((j-1+N)%N), &count, array); //left up cell
  check ( (((i-1+N)%N)*N) + ((j-1+N)%N), &count, array); //left down cell
  check ( (((i-1+N)%N)*N) + ((j+1)%N), &count, array); //right down cell
  return count;
}

__host__ __device__ void check (int index, int *count, int * array) {
  if(array[index]==1){
    (*count)++;
  }
}


void read_from_file(int *X, char *filename, int N){
  FILE *fp = fopen(filename, "r+");
  int size = fread(X, sizeof(int), N*N, fp);
  printf("elements: %d\n", size);
  fclose(fp);
}

void save_new_table(int *X, int N, int cycles){
  FILE *fp;
  char filename[20];
  sprintf(filename, "table%dx%d-%dit.bin", N, N,cycles);
  printf("Saving table in file %s\n", filename);
  fp = fopen(filename, "w+");
  fwrite(X, sizeof(int), N*N, fp);
  fclose(fp);
}










int main(int argc, char **argv){

  int *dev_in, *dev_out;
  char *filename = argv[1];
  int N = atoi(argv[2]);
  int cycles = atoi (argv[3]);
  int *table = (int *)malloc(N*N*sizeof(int));
  int l=1;
  int m=1;
  int o=1;
  int choice;
  float ttime;
  int thperblok = 4 ;
  hipError_t err;
  hipEvent_t start, stop;

  if (argc != 4) {
    printf("Usage : filename, number of elements, number of cycles\n");
    exit(1);
  }

  if ((cycles%2)!=0) {
    printf("Error : need a pair number of cycles ! Exiting...\n");
    exit(1);
  }

  printf("Enter your choice : 1 for one cell per thread, 2 for multiple cells per thread without shared memory, 3 for multiple cells per thread with shared memory, 4 for serial computation\n");
  scanf ("%d", &choice);

  if ((choice==2) || (choice==3)){
    printf("Choose a number of cells per thread so that %d/(25xnumber) is an integer\n",N);
    scanf("%d", &thperblok);
  }

  printf("Reading %dx%d table from file %s\n", N, N, filename);
  
  read_from_file(table, filename, N);



  /* Memory allocation part */
  hipMalloc( &dev_in, (N*N)*sizeof(int) );
  hipMalloc( &dev_out, (N*N)*sizeof(int) );
  
  hipMemcpy( dev_in, table, (N*N)*sizeof(int), hipMemcpyHostToDevice );

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  /*Launch of the computation*/ 
  dim3 dimBlock( BLOCKSIZE, BLOCKSIZE );

  dim3 dimGrid1= dim3(N/dimBlock.x, N/dimBlock.y);
  dim3 dimGrid2= dim3(N/(dimBlock.x*thperblok), N/dimBlock.y); //same for 2 and 3
  printf(" Size of the one per threads grid: Grid : %dx%d, blocks : %dx%d\n", dimGrid1.x, dimGrid1.y, dimBlock.x, dimBlock.y);
  printf(" Size of the multiple cells per threads grid: Grid : %dx%d, blocks : %dx%d\n", dimGrid2.x, dimGrid2.y, dimBlock.x, dimBlock.y);

  switch (choice) {
    case 1 :
    printf("Lauch of the computation with one cell per thread...\n");


    for (l=1; l<=cycles;l++){ /* Needs a pair number of cycles !*/
      if ((l%2)==1){
        game<<<dimGrid1,dimBlock>>>(N, dev_in, dev_out );
        hipDeviceSynchronize();
        hipDeviceSynchronize();
      }
      else {
            game<<<dimGrid1,dimBlock>>>(N, dev_out, dev_in ); /* Executed last, so we have the result in dev_in*/
        hipDeviceSynchronize();
        hipDeviceSynchronize();
      }
    }
    err = hipGetLastError();
    if( err != hipSuccess ) {
      printf("Error: %s\n", hipGetErrorString(err));
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ttime, start, stop);

    hipMemcpy( table, dev_in, (N*N)*sizeof(int), hipMemcpyDeviceToHost );

    save_new_table(table,N,cycles);
    printf("\nTime for 1 cell per thread= %3.1f ms\n", ttime);
    break;




    case 2:
    printf("Lauch of the computation with multiple cells per thread without shared memory...\n");
    printf("Grid : %dx%d, blocks : %dx%d", dimGrid2.x, dimGrid2.y, dimBlock.x, dimBlock.y);
    for (m=1; m<=cycles;m++){ /* Needs a pair number of cycles !*/
      if ((m%2)==1){
        game2<<<dimGrid2,dimBlock>>>(N, dev_in, dev_out,thperblok );
        hipDeviceSynchronize();
        hipDeviceSynchronize();
      }
      else {
              game2<<<dimGrid2,dimBlock>>>(N, dev_out, dev_in,thperblok ); /* Executed last, so we have the result in dev_in*/
        hipDeviceSynchronize();
        hipDeviceSynchronize();
      }
  }
err = hipGetLastError();
if( err != hipSuccess ) {
  printf("Error: %s\n", hipGetErrorString(err));
}
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipEventElapsedTime(&ttime, start, stop);

hipMemcpy( table, dev_in, (N*N)*sizeof(int), hipMemcpyDeviceToHost );
save_new_table(table,N,cycles);
printf("\nTime for %d cells per thread= %3.1f ms\n", thperblok, ttime);

break;





case 3:

printf("Lauch of the computation with multiple cells per thread with shared memory...\n");

      //game3<<<dimGrid2,dimBlock, (dimBlock.x+1)*(dimBlock.y+1)*thperblok*sizeof(int)>>>(N, dev_in, dev_out, thperblok );
     for (l=1; l<=cycles;l++){ /* Needs a pair number of cycles !*/
      if ((l%2)==1){
        game5<<<dimGrid2,dimBlock,(dimBlock.x*thperblok+2)*(dimBlock.y+2)*sizeof(int)>>>(N, dev_in, dev_out,thperblok); /*because we need all surrounding cells in the shared memory*/
        hipDeviceSynchronize();
        hipDeviceSynchronize();
      }
      else {
        game5<<<dimGrid2,dimBlock,(dimBlock.x*thperblok+2)*(dimBlock.y+2)*sizeof(int)>>>(N, dev_out, dev_in,thperblok); /* Executed last, so we have the result in dev_in*/
        hipDeviceSynchronize();
        hipDeviceSynchronize();
      }
}

err = hipGetLastError();
if( err != hipSuccess ) {
  printf("Error: %s\n", hipGetErrorString(err));
}
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipEventElapsedTime(&ttime, start, stop);

hipMemcpy( table, dev_in, (N*N)*sizeof(int), hipMemcpyDeviceToHost );
save_new_table(table,N,cycles);
printf("\nTime for %d cells per thread with shared memory= %3.1f ms\n", thperblok, ttime);
break;




case 4:
{
  clock_t begin, end;
  double time_spent;
  printf("Lauch of the serial computation...\n");
  int *newtable = (int *)malloc(N*N*sizeof(int));
  begin = clock();
  for (o=1;o<=cycles;o++){ /*Iteration loop*/
    if ((o%2)==1){
      for (l=0;l<N;l++){ /*Line indicator*/
        for(m=0;m<N;m++){ /*Row indicator*/
          int neighbours=countAlive(l,m,N,table);
              //printf("Voisins pour index %d : %d \n",l*N + m, neighbours);
          if ((neighbours==3) || ((neighbours==2) && (table[l*N + m]==1))){
            newtable [l*N + m]=1;
          }
          else {
            newtable [l*N + m]=0;
          }

        }
      }
    }
    else {
      for (l=0;l<N;l++){ /*Line indicator*/
        for(m=0;m<N;m++){/*Row indicator*/
          int neighbours=countAlive(l,m,N,newtable);
              //printf("Voisins pour index %d : %d \n",l*N + m, neighbours);
          if ((neighbours==3) || ((neighbours==2) && (newtable[l*N + m]==1))){
            table [l*N + m]=1;
          }
          else {
            table [l*N + m]=0;
          }

        }
      }
    }
  }
  end = clock();
  time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
  save_new_table(table,N,cycles);
  printf("\nTime for serial code = %3.1f ms\n",time_spent*1000);
}
break;


default :
printf("Error in the choice : must be beetwen 1 and 3. Exiting...\n");
exit(1);
}

hipFree(dev_in);
hipFree(dev_out);
free(table);
return 0;

}